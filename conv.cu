
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

class Layer
{
public:
    int M, N, O; // O: output, N: #feature, M: #params_per_feature
    float *pre_output, *output;
    float *weight, *bias;
    Layer(int M, int N, int O);
    ~Layer();
};

Layer::~Layer()
{
}

__global__ void kernel_conv_filter(float *input, float *pre_output, float *weight)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0;
    float value = 0.0;
}

__global__ void kernel_conv_bias(float *pre_output, float *bias)
{
}

__global__ void kernel_conv_sigmoid(float *pre_output, float *output)
{
    int idx = blockIdx.x * 576 + 24 * threadIdx.x + threadIdx.y;

    output[idx] = 1.0 / (1.0 + exp(-pre_output[idx]));

    printf("%f \n", output[idx]);
}

void forward_pass(double data[28][28])
{
}

int main()
{
    double data[28][28];
    forward_pass(data);
}